#include "hip/hip_runtime.h"

# include "config.h"
# include "shared.h"

# include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>

template <int N, int M> void randomDumpMatrixEle(float layer [][M], int nums, float scale) {
    srand (0);
    int total = N * M, tmp;
    for (int i=0; i < nums; i++) {
        tmp = rand() % total;
        printf(" %0.4f ", layer[tmp/M][tmp%M]*scale);
    }
    printf("\n");
}

template <int N, int M> void dumpMatrixEle(float layer [][M]) {
    for (int i=0; i < N; i++) {
        for (int j=0; j < M; j++) {
            printf(" %0.2f ", layer[i][j]);
        }
        printf("\n");
    }
}

 void dumpArray(float arr[], int n) {
    for (int i = 0; i < n; i++) {
        printf("%0.2f ", arr[i]);
    }
    printf("\n");
}

void get_mnist_grad() {
    checkCudaErrors(hipMemcpyFromSymbol(&h_g_d_fc1_w, HIP_SYMBOL(d_g_d_fc1_w), H*D*sizeof(float)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_g_d_fc1_b, HIP_SYMBOL(d_g_d_fc1_b), H*sizeof(float)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_g_d_fc2_w, HIP_SYMBOL(d_g_d_fc2_w), H*C*sizeof(float)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_g_d_fc2_b, HIP_SYMBOL(d_g_d_fc2_b), C*sizeof(float)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_loss, HIP_SYMBOL(d_loss), sizeof(float)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_count, HIP_SYMBOL(d_count), sizeof(float)));
}

void reset_mnist_grad() {
    memset(h_g_d_fc1_w, 0, H*D*sizeof(float));
    memset(h_g_d_fc1_b, 0, H*sizeof(float));
    memset(h_g_d_fc2_w, 0, C*H*sizeof(float));
    memset(h_g_d_fc2_b, 0, C*sizeof(float));
    h_loss = 0;
    h_count = 0;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_g_d_fc1_w), &h_g_d_fc1_w, H*D*sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_g_d_fc1_b), &h_g_d_fc1_b, H*sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_g_d_fc2_w), &h_g_d_fc2_w, H*C*sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_g_d_fc2_b), &h_g_d_fc2_b, C*sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_loss), &h_loss, sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_count), &h_count, sizeof(float)));
}

template <int M> __global__ void init_affine_layer_fc1(int size, hiprandState state[]) {
    int seq = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x;
    hiprand_init(1234, seq, 0, &state[seq]);
#pragma unroll
    for (int i=seq*size; i<(seq+1)*size; i++) {
        fc1[i/M][i%M] = hiprand_uniform(state+seq);
    }
}

template <int M> __global__ void init_affine_layer_fc2(int size, hiprandState state[]) {
    int seq = blockIdx.x * blockDim.x * blockDim.y + threadIdx.x;
    hiprand_init(1234, seq, 0, &state[seq]);
#pragma unroll
    for (int i=seq*size; i<(seq+1)*size; i++) {
        fc2[i/M][i%M] = hiprand_uniform(state+seq);
    }
}


// 128 * 20 * 300
void init_mnist_network() {
    hiprandState *d_state;
    checkCudaErrors(hipMalloc(&d_state, 784 * 30 * sizeof(hiprandState)));

    // init fc1 
    dim3 fc1_block_w(32, 1);
    dim3 fc1_grid_w(49, 1);
    init_affine_layer_fc1<D><<<fc1_grid_w, fc1_block_w>>>(15, d_state);
    hipDeviceSynchronize();

    memset(h_b1, 0, H * sizeof(int));
    
    // init fc2 
    dim3 fc2_block_w(H, 1);
    dim3 fc2_grid_w(C, 1);
    init_affine_layer_fc2<H><<<fc2_grid_w, fc2_block_w>>>(1, d_state);
    hipDeviceSynchronize();

    memset(h_b2, 0, C * sizeof(int));


    // sync data from gpu to cpu 
    checkCudaErrors(hipMemcpyFromSymbol(&h_fc1, HIP_SYMBOL(fc1), H * D * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b1), &h_b1, H * sizeof(float)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_fc2, HIP_SYMBOL(fc2), C * H * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b2), &h_b2, C * sizeof(float)));

    // free resource
    checkCudaErrors(hipFree(d_state));
}


void sync_mnist_model_to_gpu() {
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fc1), &h_fc1, H * D * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b1), &h_b1, H * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(fc2), &h_fc2, C * H * sizeof(float)));
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(b2), &h_b2, C * sizeof(float)));
}


template <int N, int M> void update_matrix(float dmatrix[N][M], float det[N][M], float lr, float reg) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            dmatrix[i][j] *= (1 - reg);
            dmatrix[i][j] -= lr * det[i][j]/TC ;
        }
    }
}

template <int N> void update_array(float darr[N], float det[N], float lr, float reg) {
    for (int i = 0; i < N; i++) {
        darr[i] *= (1 - reg);
        darr[i] -= lr * det[i]/TC;
    }
}

void update_mnist_model(float lr, float reg) {
    get_mnist_grad();
    printf("loss: %f, accuracy: %f\n", h_loss, float(h_count)/TNN);

    update_matrix<H, D>(h_fc1, h_g_d_fc1_w, lr, reg);
    update_array<H>(h_b1, h_g_d_fc1_b, lr, reg);
    update_matrix<C, H>(h_fc2, h_g_d_fc2_w, lr, reg);
    update_array<C>(h_b2, h_g_d_fc2_b, lr, reg);
    // randomDumpMatrixEle<C, H>(h_g_d_fc2_w, 10, lr);
    // randomDumpMatrixEle<H, D>(h_g_d_fc1_w, 10, 1.0);
    // dumpMatrixEle<C, H>(h_g_d_fc2_w);

    sync_mnist_model_to_gpu();
 
    reset_mnist_grad();
    //randomDumpMatrixEle<C, H>(h_fc2, 10, 1.0);
    //randomDumpMatrixEle<H, D>(h_fc1, 10, 1.0);

}

